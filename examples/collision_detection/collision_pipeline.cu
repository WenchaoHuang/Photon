#include "hip/hip_runtime.h"
/**
 *	Copyright (c) 2025 Wenchao Huang <physhuangwenchao@gmail.com>
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy
 *	of this software and associated documentation files (the "Software"), to deal
 *	in the Software without restriction, including without limitation the rights
 *	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *	copies of the Software, and to permit persons to whom the Software is
 *	furnished to do so, subject to the following conditions:
 *
 *	The above copyright notice and this permission notice shall be included in all
 *	copies or substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 *	SOFTWARE.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <photon/macros.h>
#include <>
#include <optix_device.h>
#include "launch_params.h"

__RT_CONSTANT__ LaunchParams launchParams;

/*********************************************************************************
*********************************    kernels    **********************************
*********************************************************************************/

__RT_KERNEL__ void __raygen__()
{
	auto vIdx = optixGetLaunchIndex().x;
	auto p0 = launchParams.vertices[vIdx];

	optixTrace(launchParams.traversable,
				float3{ p0.x, p0.y, p0.z },
				float3{ 1.0f, 0.0f, 0.0f },
				0.0f,
				1e-6f,
				0.0f,
				OptixVisibilityMask(255),
				OPTIX_RAY_FLAG_DISABLE_ANYHIT,
				0,
				0,
				0);
}


__RT_KERNEL__ void __intersection__()
{
	auto p0 = optixGetWorldRayOrigin();
	auto vIdx0 = optixGetLaunchIndex().x;
	auto vIdx1 = optixGetPrimitiveIndex();

	if (vIdx1 <= vIdx0)
		return;

	auto p1 = launchParams.vertices[vIdx1];
	float3 d = { p1.x - p0.x, p1.y - p0.y, p1.z - p0.z };
	float len = sqrtf((d.x * d.x) + (d.y * d.y) + (d.z * d.z));

	if (len < launchParams.radius)
	{
		atomicAdd(launchParams.count.data(), 1);
	}
}


__RT_KERNEL__ void __miss__()
{
//	printf("miss\n");
}