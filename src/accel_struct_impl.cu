#include "hip/hip_runtime.h"
/**
 *	Copyright (c) 2025 Wenchao Huang <physhuangwenchao@gmail.com>
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy
 *	of this software and associated documentation files (the "Software"), to deal
 *	in the Software without restriction, including without limitation the rights
 *	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *	copies of the Software, and to permit persons to whom the Software is
 *	furnished to do so, subject to the following conditions:
 *
 *	The above copyright notice and this permission notice shall be included in all
 *	copies or substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 *	SOFTWARE.
 */

#include "accel_struct_impl.h"
#include <nucleus/launch_utils.cuh>
#include <optix_stubs.h>

PHOTON_USING_NAMESPACE

/*********************************************************************************
*******************************    Validations    ********************************
*********************************************************************************/

static_assert(static_cast<int>(GeomAccelStruct::GeomFlags::None)								== OPTIX_GEOMETRY_FLAG_NONE);
static_assert(static_cast<int>(GeomAccelStruct::GeomFlags::DisableAnyhit)						== OPTIX_GEOMETRY_FLAG_DISABLE_ANYHIT);
static_assert(static_cast<int>(GeomAccelStruct::GeomFlags::RequireSingleAnyhitCall)				== OPTIX_GEOMETRY_FLAG_REQUIRE_SINGLE_ANYHIT_CALL);
#if OPTIX_VERSION >= 70500
static_assert(static_cast<int>(GeomAccelStruct::GeomFlags::DisableTriangleFaceCulling)			== OPTIX_GEOMETRY_FLAG_DISABLE_TRIANGLE_FACE_CULLING);
#endif

#if OPTIX_VERSION >= 70100
static_assert(static_cast<int>(AccelStructCurve::CurveType::RoundLinear)						== OPTIX_PRIMITIVE_TYPE_ROUND_LINEAR);
static_assert(static_cast<int>(AccelStructCurve::CurveType::RoundCubicBSpline)					== OPTIX_PRIMITIVE_TYPE_ROUND_CUBIC_BSPLINE);
static_assert(static_cast<int>(AccelStructCurve::CurveType::RoundQuadraticBSpline)				== OPTIX_PRIMITIVE_TYPE_ROUND_QUADRATIC_BSPLINE);
#endif
#if OPTIX_VERSION >= 70400
static_assert(static_cast<int>(AccelStructCurve::CurveType::RoundCatmullRom)					== OPTIX_PRIMITIVE_TYPE_ROUND_CATMULLROM);
#endif
#if OPTIX_VERSION >= 70700
static_assert(static_cast<int>(AccelStructCurve::CurveType::RoundCubicBezier)					== OPTIX_PRIMITIVE_TYPE_ROUND_CUBIC_BEZIER);
static_assert(static_cast<int>(AccelStructCurve::CurveType::FlatQuadraticBSpline)				== OPTIX_PRIMITIVE_TYPE_FLAT_QUADRATIC_BSPLINE);
#endif

static_assert(static_cast<int>(InstAccelStruct::InstFlags::None)								== OPTIX_INSTANCE_FLAG_NONE);
static_assert(static_cast<int>(InstAccelStruct::InstFlags::DisableAnyhit)						== OPTIX_INSTANCE_FLAG_DISABLE_ANYHIT);
static_assert(static_cast<int>(InstAccelStruct::InstFlags::EnforceAnyhit)						== OPTIX_INSTANCE_FLAG_ENFORCE_ANYHIT);
static_assert(static_cast<int>(InstAccelStruct::InstFlags::eFlagFlipTriangleFacing)				== OPTIX_INSTANCE_FLAG_FLIP_TRIANGLE_FACING);
static_assert(static_cast<int>(InstAccelStruct::InstFlags::DisableTriangleFaceCulling)			== OPTIX_INSTANCE_FLAG_DISABLE_TRIANGLE_FACE_CULLING);
#if OPTIX_VERSION >= 70600
static_assert(static_cast<int>(InstAccelStruct::InstFlags::DisableOpacityMicromaps)				== OPTIX_INSTANCE_FLAG_DISABLE_OPACITY_MICROMAPS);
static_assert(static_cast<int>(InstAccelStruct::InstFlags::ForceOpacityMicromapAsTwoState)		== OPTIX_INSTANCE_FLAG_FORCE_OPACITY_MICROMAP_2_STATE);
#endif

/*********************************************************************************
*****************************    AccelStructBase    ******************************
*********************************************************************************/

AccelStructBase::AccelStructBase(std::shared_ptr<DeviceContextImpl> deviceContext) : m_deviceContext(deviceContext), m_hTraversable(0), m_buildOptions(OptixAccelBuildOptions{}), m_numSbtRecords(0)
{

}


void AccelStructBase::build(ns::Stream & stream, ns::AllocPtr allocator, const std::vector<OptixBuildInput> & buildInputs, OptixAccelBuildOptions buildOptions)
{
	OptixAccelBufferSizes accelBufferSizes = {};

	buildOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

	OptixResult err = optixAccelComputeMemoryUsage(m_deviceContext->handle(), &buildOptions, buildInputs.data(), (uint32_t)buildInputs.size(), &accelBufferSizes);

	if (err == OPTIX_SUCCESS)
	{
		m_outputBuffer.resize(allocator, accelBufferSizes.outputSizeInBytes);

		m_tempBuffer.resize(allocator, NS_MAX(accelBufferSizes.tempSizeInBytes, accelBufferSizes.tempUpdateSizeInBytes) + 15);		//!	last aligned 8-bytes for storing compacted size.

		if (buildOptions.buildFlags & OPTIX_BUILD_FLAG_ALLOW_COMPACTION)
		{
			OptixAccelEmitDesc			emittedProp = {};
			OptixTraversableHandle		outputHandle = 0;
			emittedProp.type			= OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
			emittedProp.result			= hipDeviceptr_t(m_tempBuffer.data() + ((m_tempBuffer.size() - 15 + 7) / 8) * 8);

			err = optixAccelBuild(m_deviceContext->handle(), stream.handle(), &buildOptions, buildInputs.data(), (uint32_t)buildInputs.size(),
									  (hipDeviceptr_t)m_tempBuffer.data(), m_tempBuffer.bytes(), (hipDeviceptr_t)m_outputBuffer.data(), m_outputBuffer.bytes(), &outputHandle, &emittedProp, 1);

			if (err == OPTIX_SUCCESS)
			{
				uint64_t compactedSize = 0;

				stream.memcpy<void>(&compactedSize, (const void*)emittedProp.result, sizeof(uint64_t));
				stream.sync();

				m_compactedBuffer.resize(allocator, compactedSize);

				err = optixAccelCompact(m_deviceContext->handle(), stream.handle(), outputHandle, (hipDeviceptr_t)m_compactedBuffer.data(), m_compactedBuffer.bytes(), &m_hTraversable);
			}
		}
		else
		{
			err = optixAccelBuild(m_deviceContext->handle(), stream.handle(), &buildOptions, buildInputs.data(), (uint32_t)buildInputs.size(),
									  (hipDeviceptr_t)m_tempBuffer.data(), m_tempBuffer.bytes(), (hipDeviceptr_t)m_outputBuffer.data(), m_outputBuffer.bytes(), &m_hTraversable, nullptr, 0);
		}
	}

	if (err != OPTIX_SUCCESS)
	{
		NS_ERROR_LOG("Failed to build acceleration structure: %s.", optixGetErrorString(err));

		throw err;
	}

	m_buildOptions = buildOptions;
	m_buildInputs = buildInputs;
}


void AccelStructBase::rebuild(ns::Stream & stream)
{
	if (m_hTraversable != 0)
	{
		OptixTraversableHandle outputHandle = 0;

		m_buildOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

		OptixResult err = optixAccelBuild(m_deviceContext->handle(), stream.handle(), &m_buildOptions, m_buildInputs.data(), (uint32_t)m_buildInputs.size(),
											  (hipDeviceptr_t)m_tempBuffer.data(), m_tempBuffer.bytes(), (hipDeviceptr_t)m_outputBuffer.data(), m_outputBuffer.bytes(), &outputHandle, nullptr, 0);

		if (err == OPTIX_SUCCESS)
		{
			if (this->allowCompaction())
			{
				err = optixAccelCompact(m_deviceContext->handle(), stream.handle(), outputHandle, (hipDeviceptr_t)m_compactedBuffer.data(), m_compactedBuffer.bytes(), &m_hTraversable);
			}
			else
			{
				m_hTraversable = outputHandle;
			}
		}

		if (err != OPTIX_SUCCESS)
		{
			NS_ERROR_LOG("Failed to rebuild acceleration structure: %s.", optixGetErrorString(err));

			throw err;
		}
	}
}


void AccelStructBase::refit(ns::Stream & stream)
{
	OptixResult err = OPTIX_SUCCESS;

	if (this->allowUpdate() && (m_hTraversable != 0))
	{
		m_buildOptions.operation = OPTIX_BUILD_OPERATION_UPDATE;

		if (this->allowCompaction())
		{
			err = optixAccelBuild(m_deviceContext->handle(), stream.handle(), &m_buildOptions, m_buildInputs.data(), (uint32_t)m_buildInputs.size(),
									  (hipDeviceptr_t)m_tempBuffer.data(), m_tempBuffer.bytes(), (hipDeviceptr_t)m_compactedBuffer.data(), m_compactedBuffer.bytes(), &m_hTraversable, nullptr, 0);
		}
		else
		{
			err = optixAccelBuild(m_deviceContext->handle(), stream.handle(), &m_buildOptions, m_buildInputs.data(), (uint32_t)m_buildInputs.size(),
									  (hipDeviceptr_t)m_tempBuffer.data(), m_tempBuffer.bytes(), (hipDeviceptr_t)m_outputBuffer.data(), m_outputBuffer.bytes(), &m_hTraversable, nullptr, 0);
		}

		if (err != OPTIX_SUCCESS)
		{
			NS_ERROR_LOG("Failed to refit acceleration structure: %s.", optixGetErrorString(err));

			throw err;
		}
	}
	else if (!this->allowUpdate())
	{
		NS_WARNING_LOG("Acceleration structure is non-updatable!");
	}
}


AccelStructBase::~AccelStructBase()
{

}

/*********************************************************************************
*************************    AccelStructTriangleImpl    **************************
*********************************************************************************/

void AccelStructTriangleImpl::build(ns::Stream & stream, ns::AllocPtr allocator, ns::ArrayProxy<BuildInput> buildInputs, bool preferFastTrace, bool allowUpdate)
{
	m_numSbtRecords = 0;
	m_geomFlags.resize(buildInputs.size());
	m_vertBuffers.resize(buildInputs.size());
	m_buildInputs.resize(buildInputs.size());

	std::vector<OptixBuildInput> optixBuildInputs(buildInputs.size());

	for (size_t i = 0; i < optixBuildInputs.size(); i++)
	{
		const bool useInexBuffer = (buildInputs[i].indexBuffer != nullptr) && (buildInputs[i].numIndexTriplets > 0);

		if (buildInputs[i].perSbtRecordFlags.empty())
		{
			m_geomFlags[i].assign(buildInputs[i].numSbtRecords, OPTIX_GEOMETRY_FLAG_NONE);
		}
		else if (buildInputs[i].perSbtRecordFlags.size() == buildInputs[i].numSbtRecords)
		{
			m_geomFlags[i].resize(buildInputs[i].perSbtRecordFlags.size());

			std::memcpy(m_geomFlags[i].data(),buildInputs[i].perSbtRecordFlags.data(), sizeof(GeomFlags) * buildInputs[i].perSbtRecordFlags.size());
		}
		else
		{
			NS_ASSERT_LOG_IF(buildInputs[i].perSbtRecordFlags.size() != buildInputs[i].numSbtRecords, "Geometry flags does not match with numSbtRecords!");

			return;
		}

		m_buildInputs[i]													= buildInputs[i];
		m_vertBuffers[i]													= (hipDeviceptr_t)buildInputs[i].vertexBuffer.data();
		m_numSbtRecords														+= buildInputs[i].numSbtRecords;
		optixBuildInputs[i].type											= OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
		optixBuildInputs[i].triangleArray.flags								= m_geomFlags[i].data();
		optixBuildInputs[i].triangleArray.vertexFormat						= OPTIX_VERTEX_FORMAT_FLOAT3;
		optixBuildInputs[i].triangleArray.vertexStrideInBytes				= sizeof(ns::float3_16a);
		optixBuildInputs[i].triangleArray.vertexBuffers						= &m_vertBuffers[i];
		optixBuildInputs[i].triangleArray.numVertices						= buildInputs[i].numVertices;
		optixBuildInputs[i].triangleArray.indexBuffer						= useInexBuffer ? (hipDeviceptr_t)buildInputs[i].indexBuffer.data() : NULL;
		optixBuildInputs[i].triangleArray.numIndexTriplets					= useInexBuffer ? buildInputs[i].numIndexTriplets : 0;
		optixBuildInputs[i].triangleArray.indexStrideInBytes				= useInexBuffer ? sizeof(ns::int3_16a) : 0;
		optixBuildInputs[i].triangleArray.preTransform						= NULL;
		optixBuildInputs[i].triangleArray.numSbtRecords						= buildInputs[i].numSbtRecords;
		optixBuildInputs[i].triangleArray.primitiveIndexOffset				= 0;
		optixBuildInputs[i].triangleArray.sbtIndexOffsetBuffer				= (hipDeviceptr_t)buildInputs[i].sbtIndexOffsetBuffer.data();
		optixBuildInputs[i].triangleArray.sbtIndexOffsetSizeInBytes			= sizeof(uint32_t);
		optixBuildInputs[i].triangleArray.sbtIndexOffsetStrideInBytes		= sizeof(uint32_t);
	#if OPTIX_VERSION >= 70100
		optixBuildInputs[i].triangleArray.indexFormat						= useInexBuffer ? OPTIX_INDICES_FORMAT_UNSIGNED_INT3 : OPTIX_INDICES_FORMAT_NONE;
		optixBuildInputs[i].triangleArray.transformFormat					= OPTIX_TRANSFORM_FORMAT_NONE;
	#else
		optixBuildInputs[i].triangleArray.indexFormat						= OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
	#endif
	}

	OptixAccelBuildOptions						buildOptions = {};
	buildOptions.operation						= OPTIX_BUILD_OPERATION_BUILD;
	buildOptions.buildFlags						= OPTIX_BUILD_FLAG_NONE;
//	buildOptions.buildFlags						|= OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	buildOptions.buildFlags						|= preferFastTrace ? OPTIX_BUILD_FLAG_PREFER_FAST_TRACE : OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
	buildOptions.buildFlags						|= allowUpdate ? OPTIX_BUILD_FLAG_ALLOW_UPDATE : 0;
	buildOptions.motionOptions.numKeys			= 0;
	buildOptions.motionOptions.timeBegin		= 0.0f;
	buildOptions.motionOptions.timeEnd			= 0.0f;
	buildOptions.motionOptions.flags			= OPTIX_MOTION_FLAG_NONE;

	AccelStructBase::build(stream, allocator, optixBuildInputs, buildOptions);
}

/*********************************************************************************
***************************    AccelStructAabbImpl    ****************************
*********************************************************************************/

void AccelStructAabbImpl::build(ns::Stream & stream, ns::AllocPtr allocator, ns::ArrayProxy<BuildInput> buildInputs, bool preferFastTrace, bool allowUpdate)
{
	m_numSbtRecords = 0;
	m_geomFlags.resize(buildInputs.size());
	m_aabbBuffers.resize(buildInputs.size());
	m_buildInputs.resize(buildInputs.size());

	std::vector<OptixBuildInput> optixBuildInputs(buildInputs.size());

	for (size_t i = 0; i < optixBuildInputs.size(); i++)
	{
		if (buildInputs[i].perSbtRecordFlags.empty())
		{
			m_geomFlags[i].assign(buildInputs[i].numSbtRecords, OPTIX_GEOMETRY_FLAG_NONE);
		}
		else if (buildInputs[i].perSbtRecordFlags.size() == buildInputs[i].numSbtRecords)
		{
			m_geomFlags[i].resize(buildInputs[i].perSbtRecordFlags.size());

			std::memcpy(m_geomFlags[i].data(),buildInputs[i].perSbtRecordFlags.data(), sizeof(GeomFlags) * buildInputs[i].perSbtRecordFlags.size());
		}
		else
		{
			NS_ASSERT_LOG_IF(buildInputs[i].perSbtRecordFlags.size() != buildInputs[i].numSbtRecords, "Geometry flags does not match with numSbtRecords!");

			return;
		}

		m_buildInputs[i]															= buildInputs[i];
		m_aabbBuffers[i]															= (hipDeviceptr_t)buildInputs[i].aabbBuffer.data();
		m_numSbtRecords																+= buildInputs[i].numSbtRecords;
		optixBuildInputs[i].type													= OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
	#if OPTIX_VERSION >= 70100
		optixBuildInputs[i].customPrimitiveArray.flags								= m_geomFlags[i].data();
		optixBuildInputs[i].customPrimitiveArray.aabbBuffers						= &m_aabbBuffers[i];
		optixBuildInputs[i].customPrimitiveArray.strideInBytes						= sizeof(Aabb);
		optixBuildInputs[i].customPrimitiveArray.numPrimitives						= buildInputs[i].numPrimitives;
		optixBuildInputs[i].customPrimitiveArray.numSbtRecords						= buildInputs[i].numSbtRecords;
		optixBuildInputs[i].customPrimitiveArray.primitiveIndexOffset				= 0;
		optixBuildInputs[i].customPrimitiveArray.sbtIndexOffsetBuffer				= (hipDeviceptr_t)buildInputs[i].sbtIndexOffsetBuffer.data();
		optixBuildInputs[i].customPrimitiveArray.sbtIndexOffsetSizeInBytes			= sizeof(uint32_t);
		optixBuildInputs[i].customPrimitiveArray.sbtIndexOffsetStrideInBytes		= sizeof(uint32_t);
	#else
		optixBuildInputs[i].aabbArray.flags											= m_geomFlags[i].data();
		optixBuildInputs[i].aabbArray.aabbBuffers									= &m_aabbBuffers[i];
		optixBuildInputs[i].aabbArray.strideInBytes									= sizeof(Aabb);
		optixBuildInputs[i].aabbArray.numPrimitives									= buildInputs[i].numPrimitives;
		optixBuildInputs[i].aabbArray.numSbtRecords									= buildInputs[i].numSbtRecords;
		optixBuildInputs[i].aabbArray.primitiveIndexOffset							= 0;
		optixBuildInputs[i].aabbArray.sbtIndexOffsetBuffer							= (hipDeviceptr_t)buildInputs[i].sbtIndexOffsetBuffer.data();
		optixBuildInputs[i].aabbArray.sbtIndexOffsetSizeInBytes						= sizeof(uint32_t);
		optixBuildInputs[i].aabbArray.sbtIndexOffsetStrideInBytes					= sizeof(uint32_t);
	#endif
	}

	OptixAccelBuildOptions						buildOptions = {};
	buildOptions.operation						= OPTIX_BUILD_OPERATION_BUILD;
	buildOptions.buildFlags						= OPTIX_BUILD_FLAG_NONE;
//	buildOptions.buildFlags						|= OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	buildOptions.buildFlags						|= preferFastTrace ? OPTIX_BUILD_FLAG_PREFER_FAST_TRACE : OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
	buildOptions.buildFlags						|= allowUpdate ? OPTIX_BUILD_FLAG_ALLOW_UPDATE : 0;
	buildOptions.motionOptions.numKeys			= 0;
	buildOptions.motionOptions.timeBegin		= 0.0f;
	buildOptions.motionOptions.timeEnd			= 0.0f;
	buildOptions.motionOptions.flags			= OPTIX_MOTION_FLAG_NONE;

	AccelStructBase::build(stream, allocator, optixBuildInputs, buildOptions);
}

/*********************************************************************************
***************************    AccelStructCurveImpl    ***************************
*********************************************************************************/

void AccelStructCurveImpl::build(ns::Stream & stream, ns::AllocPtr allocator, ns::ArrayProxy<BuildInput> buildInputs, bool preferFastTrace, bool allowUpdate)
{
	m_buildInputs.resize(buildInputs.size());
	m_vertBuffers.resize(buildInputs.size());
	m_widthBuffers.resize(buildInputs.size());
	m_numSbtRecords = static_cast<uint32_t>(buildInputs.size());

	std::vector<OptixBuildInput> optixBuildInputs(buildInputs.size());

	for (size_t i = 0; i < buildInputs.size(); i++)
	{
		m_buildInputs[i]										= buildInputs[i];
		m_vertBuffers[i]										= (hipDeviceptr_t)buildInputs[i].vertexBuffer.data();
		m_widthBuffers[i]										= (hipDeviceptr_t)buildInputs[i].widthBuffer.data();

	#if OPTIX_VERSION >= 70400
		optixBuildInputs[i].curveArray.endcapFlags				= OPTIX_CURVE_ENDCAP_DEFAULT;
	#endif
	#if OPTIX_VERSION >= 70100
		optixBuildInputs[i].type								= OPTIX_BUILD_INPUT_TYPE_CURVES;
		optixBuildInputs[i].curveArray.flag						= buildInputs[i].flags;
		optixBuildInputs[i].curveArray.curveType				= static_cast<OptixPrimitiveType>(buildInputs[i].curveType);
		optixBuildInputs[i].curveArray.numVertices				= buildInputs[i].numVertices;
		optixBuildInputs[i].curveArray.numPrimitives			= buildInputs[i].numPrimitives;
		optixBuildInputs[i].curveArray.vertexBuffers			= &m_vertBuffers[i];
		optixBuildInputs[i].curveArray.vertexStrideInBytes		= sizeof(ns::float3_16a);
		optixBuildInputs[i].curveArray.indexBuffer				= (hipDeviceptr_t)buildInputs[i].indexBuffer.data();
		optixBuildInputs[i].curveArray.indexStrideInBytes		= sizeof(uint32_t);
		optixBuildInputs[i].curveArray.widthBuffers				= &m_widthBuffers[i];
		optixBuildInputs[i].curveArray.widthStrideInBytes		= sizeof(float);
		optixBuildInputs[i].curveArray.normalBuffers			= nullptr;
		optixBuildInputs[i].curveArray.normalStrideInBytes		= 0;
		optixBuildInputs[i].curveArray.primitiveIndexOffset		= 0;
	#endif
	}

	OptixAccelBuildOptions						buildOptions = {};
	buildOptions.operation						= OPTIX_BUILD_OPERATION_BUILD;
	buildOptions.buildFlags						= OPTIX_BUILD_FLAG_NONE;
//	buildOptions.buildFlags						|= OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	buildOptions.buildFlags						|= preferFastTrace ? OPTIX_BUILD_FLAG_PREFER_FAST_TRACE : OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
	buildOptions.buildFlags						|= allowUpdate ? OPTIX_BUILD_FLAG_ALLOW_UPDATE : 0;
	buildOptions.motionOptions.numKeys			= 0;
	buildOptions.motionOptions.timeBegin		= 0.0f;
	buildOptions.motionOptions.timeEnd			= 0.0f;
	buildOptions.motionOptions.flags			= OPTIX_MOTION_FLAG_NONE;

	AccelStructBase::build(stream, allocator, optixBuildInputs, buildOptions);
}

/*********************************************************************************
**************************    AccelStructSphereImpl    ***************************
*********************************************************************************/

void AccelStructSphereImpl::build(ns::Stream & stream, ns::AllocPtr allocator, ns::ArrayProxy<BuildInput> buildInputs, bool preferFastTrace, bool allowUpdate)
{
	m_numSbtRecords = 0;
	m_geomFlags.resize(buildInputs.size());
	m_buildInputs.resize(buildInputs.size());
	m_vertBuffers.resize(buildInputs.size());
	m_radiusBuffers.resize(buildInputs.size());

	std::vector<OptixBuildInput> optixBuildInputs(buildInputs.size());
	
	for (size_t i = 0; i < buildInputs.size(); i++)
	{
		if (buildInputs[i].perSbtRecordFlags.empty())
		{
			m_geomFlags[i].assign(buildInputs[i].numSbtRecords, OPTIX_GEOMETRY_FLAG_NONE);
		}
		else if (buildInputs[i].perSbtRecordFlags.size() == buildInputs[i].numSbtRecords)
		{
			m_geomFlags[i].resize(buildInputs[i].perSbtRecordFlags.size());

			std::memcpy(m_geomFlags[i].data(),buildInputs[i].perSbtRecordFlags.data(), sizeof(GeomFlags) * buildInputs[i].perSbtRecordFlags.size());
		}
		else
		{
			NS_ASSERT_LOG_IF(buildInputs[i].perSbtRecordFlags.size() != buildInputs[i].numSbtRecords, "Geometry flags does not match with numSbtRecords!");

			return;
		}

		m_buildInputs[i]												= buildInputs[i];
		m_vertBuffers[i]												= (hipDeviceptr_t)buildInputs[i].vertexBuffer.data();
		m_radiusBuffers[i]												= (hipDeviceptr_t)buildInputs[i].radiusBuffer.data();
		m_numSbtRecords													+= buildInputs[i].numSbtRecords;
	#if OPTIX_VERSION >= 70500
		optixBuildInputs[i].type										= OPTIX_BUILD_INPUT_TYPE_SPHERES;
		optixBuildInputs[i].sphereArray.flags							= m_geomFlags[i].data();
		optixBuildInputs[i].sphereArray.numVertices						= buildInputs[i].numVertices;
		optixBuildInputs[i].sphereArray.vertexBuffers					= &m_vertBuffers[i];
		optixBuildInputs[i].sphereArray.radiusBuffers					= &m_radiusBuffers[i];
		optixBuildInputs[i].sphereArray.radiusStrideInBytes				= sizeof(float);
		optixBuildInputs[i].sphereArray.singleRadius					= buildInputs[i].singleRadius;
		optixBuildInputs[i].sphereArray.numSbtRecords					= buildInputs[i].numSbtRecords;
		optixBuildInputs[i].sphereArray.primitiveIndexOffset			= 0;
		optixBuildInputs[i].sphereArray.sbtIndexOffsetBuffer			= (hipDeviceptr_t)buildInputs[i].sbtIndexOffsetBuffer.data();
		optixBuildInputs[i].sphereArray.sbtIndexOffsetSizeInBytes		= sizeof(uint32_t);
		optixBuildInputs[i].sphereArray.sbtIndexOffsetStrideInBytes		= sizeof(uint32_t);
	#endif
	}

	OptixAccelBuildOptions						buildOptions = {};
	buildOptions.operation						= OPTIX_BUILD_OPERATION_BUILD;
	buildOptions.buildFlags						= OPTIX_BUILD_FLAG_NONE;
//	buildOptions.buildFlags						|= OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	buildOptions.buildFlags						|= preferFastTrace ? OPTIX_BUILD_FLAG_PREFER_FAST_TRACE : OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
	buildOptions.buildFlags						|= allowUpdate ? OPTIX_BUILD_FLAG_ALLOW_UPDATE : 0;
	buildOptions.motionOptions.numKeys			= 0;
	buildOptions.motionOptions.timeBegin		= 0.0f;
	buildOptions.motionOptions.timeEnd			= 0.0f;
	buildOptions.motionOptions.flags			= OPTIX_MOTION_FLAG_NONE;

	AccelStructBase::build(stream, allocator, optixBuildInputs, buildOptions);
}

/*********************************************************************************
***************************    InstAccelStructImpl    ****************************
*********************************************************************************/

namespace kernels
{
	__global__ void AssignInstanceTransforms(dev::Ptr<OptixInstance> pInstances, dev::Ptr<const dev::Ptr<const Mat4x4>> ppTransforms, unsigned int numInstances)
	{
		CUDA_for(i, numInstances);

		Mat4x4 transform = {};
		transform.rows[0] = ns::float4{ 1, 0, 0, 0 };
		transform.rows[1] = ns::float4{ 0, 1, 0, 0 };
		transform.rows[2] = ns::float4{ 0, 0, 1, 0 };
		transform.rows[3] = ns::float4{ 0, 0, 0, 1 };

		if (ppTransforms[i])
		{
			transform = *ppTransforms[i];
		}

		ns::float4 * pAddressBegin = reinterpret_cast<ns::float4*>(pInstances[i].transform);

		pAddressBegin[0] = transform.rows[0];
		pAddressBegin[1] = transform.rows[1];
		pAddressBegin[2] = transform.rows[2];
	}
}

void InstAccelStructImpl::build(ns::Stream & stream, ns::AllocPtr allocator, ns::ArrayProxy<BuildInput> buildInputs, bool preferFastTrace, bool allowUpdate)
{
	m_buildInputs.resize(buildInputs.size());
	m_instances.resize(allocator, buildInputs.size());
	m_transforms.resize(allocator, buildInputs.size());

	std::vector<OptixInstance>					instances(buildInputs.size());
	std::vector<ns::dev::Ptr<const Mat4x4>>		pTransforms(buildInputs.size());

	for (size_t i = 0; i < buildInputs.size(); i++)
	{
		instances[i]						= OptixInstance{};
		instances[i].traversableHandle		= buildInputs[i].geomAccelStruct->handle();
		instances[i].visibilityMask			= buildInputs[i].visibilityMask;
		instances[i].instanceId				= buildInputs[i].instanceId;
		instances[i].sbtOffset				= buildInputs[i].sbtOffset;
		instances[i].flags					= buildInputs[i].flags;
		pTransforms[i]						= buildInputs[i].transform;
		m_buildInputs[i]					= buildInputs[i];
	}

	stream.memcpy(m_instances.data(), instances.data(), instances.size());
	stream.memcpy(m_transforms.data(), pTransforms.data(), pTransforms.size());
	stream.launch(kernels::AssignInstanceTransforms, ns::ceil_div(m_instances.size(), 128), 128)(m_instances, m_transforms, static_cast<uint32_t>(m_instances.size()));

	OptixBuildInput										optixBuildInput = {};
	optixBuildInput.type								= OPTIX_BUILD_INPUT_TYPE_INSTANCES;
	optixBuildInput.instanceArray.instances				= (hipDeviceptr_t)m_instances.data();
#if OPTIX_VERSION >= 70600
	optixBuildInput.instanceArray.instanceStride		= sizeof(OptixInstance);
#endif
	optixBuildInput.instanceArray.numInstances			= static_cast<uint32_t>(m_instances.size());

	OptixAccelBuildOptions								buildOptions = {};
	buildOptions.operation								= OPTIX_BUILD_OPERATION_BUILD;
	buildOptions.buildFlags								= OPTIX_BUILD_FLAG_NONE;
//	buildOptions.buildFlags								|= OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	buildOptions.buildFlags								|= preferFastTrace ? OPTIX_BUILD_FLAG_PREFER_FAST_TRACE : OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
	buildOptions.buildFlags								|= allowUpdate ? OPTIX_BUILD_FLAG_ALLOW_UPDATE : 0;
	buildOptions.motionOptions.numKeys					= 0;
	buildOptions.motionOptions.timeBegin				= 0.0f;
	buildOptions.motionOptions.timeEnd					= 0.0f;
	buildOptions.motionOptions.flags					= OPTIX_MOTION_FLAG_NONE;

	AccelStructBase::build(stream, allocator, { optixBuildInput }, buildOptions);
}


void InstAccelStructImpl::rebuild(ns::Stream & stream)
{
	stream.launch(kernels::AssignInstanceTransforms, ns::ceil_div(m_instances.size(), 128), 128)(m_instances, m_transforms, static_cast<uint32_t>(m_instances.size()));

	AccelStructBase::rebuild(stream);
}


void InstAccelStructImpl::refit(ns::Stream & stream)
{
	stream.launch(kernels::AssignInstanceTransforms, ns::ceil_div(m_instances.size(), 128), 128)(m_instances, m_transforms, static_cast<uint32_t>(m_instances.size()));

	AccelStructBase::refit(stream);
}