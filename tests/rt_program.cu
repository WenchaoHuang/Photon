#include "hip/hip_runtime.h"
/**
 *	Copyright (c) 2025 Wenchao Huang <physhuangwenchao@gmail.com>
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy
 *	of this software and associated documentation files (the "Software"), to deal
 *	in the Software without restriction, including without limitation the rights
 *	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *	copies of the Software, and to permit persons to whom the Software is
 *	furnished to do so, subject to the following conditions:
 *
 *	The above copyright notice and this permission notice shall be included in all
 *	copies or substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 *	SOFTWARE.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include "../deps/optix-dev/include/optix_device.h"
#include "launch_params.h"

static __constant__ LaunchParams launchParams;

extern "C"
{
	__global__ void __raygen__()
	{
		int seed = launchParams.seed;

		printf("seed[%d] = %d\n", optixGetLaunchIndex().x, seed);
	}

	__global__ void __anyhit__()
	{

	}

	__global__ void __closesthit__()
	{

	}

	__global__ void __intersection__()
	{

	}

	__global__ void __direct_callable__()
	{

	}

	__global__ void __continuation_callable__()
	{

	}

	__global__ void __exception__()
	{

	}

	__global__ void __miss__()
	{

	}
}