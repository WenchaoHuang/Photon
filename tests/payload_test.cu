#include "hip/hip_runtime.h"
/**
 *	Copyright (c) 2025 Wenchao Huang <physhuangwenchao@gmail.com>
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy
 *	of this software and associated documentation files (the "Software"), to deal
 *	in the Software without restriction, including without limitation the rights
 *	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *	copies of the Software, and to permit persons to whom the Software is
 *	furnished to do so, subject to the following conditions:
 *
 *	The above copyright notice and this permission notice shall be included in all
 *	copies or substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 *	SOFTWARE.
 */
#pragma once

#include <photon/payload.cuh>

using RayDirType = pt::Payload<float3, 0, 1, 2>;

/*********************************************************************************
*********************************    kernels    **********************************
*********************************************************************************/

__RT_KERNEL__ void __raygen__()
{
	RayDirType rayDir;
	pt::setPayload<0>(0);
	pt::setPayload<long long, 0, 1>(0);
	pt::setPayload(rayDir);
}


__RT_KERNEL__ void __miss__()
{
	pt::setPayload<0>(0);
	auto p0 = pt::getPayload<0>();
	auto p1 = pt::getPayload<float, 0>();
	auto rayDir = pt::getPayload<RayDirType>();
	rayDir = float3{ 0, 1, 0 };
	pt::setPayload(rayDir);
}